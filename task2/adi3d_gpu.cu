#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <assert.h>
#include <fstream>
#include <string>

struct KernelStats {
    float x_sweep_time = 0;
    float y_sweep_time = 0;
    float to_z_major_time = 0;
    float z_sweep_time = 0;
    float from_z_major_time = 0;
    float total_time = 0;
    int block_x = 0;
    int block_y = 0;
};

__device__ clock_t kernel_start_time;
__device__ clock_t kernel_end_time;

#ifdef USE_REAL_DOUBLE
typedef double real_t;
#else
typedef float real_t;
#endif

#define MAXEPS 0.01
#define EPS_FORMAT "%14.7E"

#ifndef NX
#define NX 900
#endif

#ifndef NY
#define NY 900
#endif

#ifndef NZ
#define NZ 900
#endif

#ifndef ITMAX
#define ITMAX 10
#endif

const int DEFAULT_BLOCK_X = 16;
const int DEFAULT_BLOCK_Y = 16;

#define CUDA_CHECK(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

template<typename T>
__device__ static T MyatomicMax(T* address, T val) {
    if (sizeof(T) == sizeof(float)) {
        int* address_as_int = (int*)address;
        int vl1 = *address_as_int, vl2;
        do {
            vl2 = vl1;
            vl1 = atomicCAS(address_as_int, vl2,
                __float_as_int(fmaxf(val, __int_as_float(vl2))));
        } while (vl2 != vl1);
        return __int_as_float(vl1);
    } else if (sizeof(T) == sizeof(double)) {
        unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
        unsigned long long int vl1 = *address_as_ull, vl2;
        do {
            vl2 = vl1;
            vl1 = atomicCAS(address_as_ull, vl2,
                __double_as_longlong(fmax(val, __longlong_as_double(vl2))));
        } while (vl2 != vl1);
        return __longlong_as_double(vl1);
    }
    return 0;
}

__global__ void x_sweep(real_t* a, int nx, int ny, int nz) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= 1 && j < ny-1 && k >= 1 && k < nz-1) {
        for (int i = 1; i < nx-1; i++) {
            int idx = i * ny * nz + j * nz + k;
            int idx_prev = (i-1) * ny * nz + j * nz + k;
            int idx_next = (i+1) * ny * nz + j * nz + k;

            a[idx] = (a[idx_prev] + a[idx_next]) / 2;
        }
    }
}

__global__ void y_sweep(real_t* a, int nx, int ny, int nz) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= 1 && i < nx-1 && k >= 1 && k < nz-1) {
        for (int j = 1; j < ny-1; j++) {
            int idx = i * ny * nz + j * nz + k;
            int idx_prev = i * ny * nz + (j-1) * nz + k;
            int idx_next = i * ny * nz + (j+1) * nz + k;

            a[idx] = (a[idx_prev] + a[idx_next]) / 2;
        }
    }
}

__global__ void reorder_to_z_major(real_t* a, real_t* b, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        for (int k = 0; k < nz; k++) {
            int idx_a = k * ny * nz + i * nz + j;         // [i][j][k]
            int idx_b = j * nx * ny + k * ny + i;         // [k][i][j]
            b[idx_b] = a[idx_a];
        }
    }
}

__global__ void z_sweep(real_t* b, int nx, int ny, int nz, real_t* d_eps) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    real_t local_eps = 0;

    if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1) {
        for (int k = 1; k < nz - 1; k++) {
            int idx      = k * nx * ny + i * ny + j;     // b[k][i][j]
            int idx_prev = (k - 1) * nx * ny + i * ny + j;
            int idx_next = (k + 1) * nx * ny + i * ny + j;

            real_t tmp1 = (b[idx_prev] + b[idx_next]) / 2;
            real_t tmp2 = fabs(b[idx] - tmp1);
            local_eps = max(local_eps, tmp2);
            b[idx] = tmp1;
        }
    }

    extern __shared__ real_t shared_eps[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    shared_eps[tid] = local_eps;
    __syncthreads();

    for (int s = blockDim.x * blockDim.y  / 2; s > 0; s /= 2) {
        if (tid < s) {
            shared_eps[tid] = max(shared_eps[tid], shared_eps[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        MyatomicMax(d_eps, shared_eps[0]);
    }
}

__global__ void reorder_from_z_major(real_t* b, real_t* a, int nx, int ny, int nz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        for (int k = 0; k < nz; k++) {
            int idx_b = i * nx * ny + k * ny + j;         // [k][i][j]
            int idx_a = k * ny * nz + j * nz + i;         // [i][j][k]
            a[idx_a] = b[idx_b];
        }
    }
}

void init(real_t *a, int nx, int ny, int nz) {
    for (int i = 0; i < nx; i++)
        for (int j = 0; j < ny; j++)
            for (int k = 0; k < nz; k++) {
                int idx = i * ny * nz + j * nz + k;
                if (k == 0 || k == nz - 1 || j == 0 || j == ny - 1 || i == 0 || i == nx - 1)
                    a[idx] = (real_t)10.0 * i / (nx - 1) + (real_t)10.0 * j / (ny - 1) + (real_t)10.0 * k / (nz - 1);
                else
                    a[idx] = (real_t)0;
            }
}

void print_gpu_info() {
    hipDeviceProp_t prop;
    int device;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    
    printf("GPU Device: %s\n", prop.name);
    printf("Total Global Memory: %.2f MB\n", prop.totalGlobalMem / (1024.0 * 1024.0));
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Dim: %d x %d x %d\n", 
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
}

void write_kernel_stats(const KernelStats& stats, const std::string& filename) {
    std::ofstream out(filename, std::ios::app);
    if (!out.is_open()) {
        printf("Failed to open profile file\n");
        return;
    }
    
    out << "Block size: " << stats.block_x << "x" << stats.block_y << "\n";
    out << "x_sweep: " << stats.x_sweep_time << " ms\n";
    out << "y_sweep: " << stats.y_sweep_time << " ms\n";
    out << "to_z_major: " << stats.to_z_major_time << " ms\n";
    out << "z_sweep: " << stats.z_sweep_time << " ms\n";
    out << "from_z_major: " << stats.from_z_major_time << " ms\n";
    out << "Total: " << stats.total_time << " ms\n\n";
    out.close();
}

int main(int argc, char** argv) {
    int block_x = DEFAULT_BLOCK_X;
    int block_y = DEFAULT_BLOCK_Y;
    
    if (argc == 3) {
        block_x = atoi(argv[1]);
        block_y = atoi(argv[2]);
    }
    
    dim3 blockSize(block_x, block_y);
    
    print_gpu_info();
    
    int nx = NX, ny = NY, nz = NZ;
    printf("Using array size: %d x %d x %d\n", nx, ny, nz);
    printf("Memory used per array: %.2f MB\n", (nx*ny*nz*sizeof(real_t)) / (1024.0*1024.0));
    
    real_t *h_A = (real_t*)malloc(nx * ny * nz * sizeof(real_t));
    if (!h_A) {
        printf("Host memory allocation failed\n");
        return 1;
    }
    
    init(h_A, nx, ny, nz);
    
    real_t *d_A, *d_B, *d_eps;
    CUDA_CHECK(hipMalloc(&d_A, nx * ny * nz * sizeof(real_t)));
    CUDA_CHECK(hipMalloc(&d_B, nx * ny * nz * sizeof(real_t)));
    CUDA_CHECK(hipMalloc(&d_eps, sizeof(real_t)));
    
    CUDA_CHECK(hipMemcpy(d_A, h_A, nx * ny * nz * sizeof(real_t), hipMemcpyHostToDevice));
    
    dim3 gridSizeX((ny + blockSize.x - 1) / blockSize.x, (nz + blockSize.y - 1) / blockSize.y);
    dim3 gridSizeY((nx + blockSize.x - 1) / blockSize.x, (nz + blockSize.y - 1) / blockSize.y);
    dim3 gridSizeZ((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y - 1) / blockSize.y);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));
    
    KernelStats stats;
    stats.block_x = block_x;
    stats.block_y = block_y;
    
    hipEvent_t start_kernel, stop_kernel;
    CUDA_CHECK(hipEventCreate(&start_kernel));
    CUDA_CHECK(hipEventCreate(&stop_kernel));
    
    for (int it = 1; it <= ITMAX; it++) {
        real_t h_eps = 0;
        CUDA_CHECK(hipMemcpy(d_eps, &h_eps, sizeof(real_t), hipMemcpyHostToDevice));
        
        // x_sweep
        CUDA_CHECK(hipEventRecord(start_kernel));
        x_sweep<<<gridSizeX, blockSize>>>(d_A, nx, ny, nz);
        CUDA_CHECK(hipEventRecord(stop_kernel));
        CUDA_CHECK(hipEventSynchronize(stop_kernel));
        CUDA_CHECK(hipEventElapsedTime(&stats.x_sweep_time, start_kernel, stop_kernel));
        
        // y_sweep
        CUDA_CHECK(hipEventRecord(start_kernel));
        y_sweep<<<gridSizeY, blockSize>>>(d_A, nx, ny, nz);
        CUDA_CHECK(hipEventRecord(stop_kernel));
        CUDA_CHECK(hipEventSynchronize(stop_kernel));
        CUDA_CHECK(hipEventElapsedTime(&stats.y_sweep_time, start_kernel, stop_kernel));
        
        // reorder_to_z_major
        CUDA_CHECK(hipEventRecord(start_kernel));
        reorder_to_z_major<<<gridSizeZ, blockSize>>>(d_A, d_B, nx, ny, nz);
        CUDA_CHECK(hipEventRecord(stop_kernel));
        CUDA_CHECK(hipEventSynchronize(stop_kernel));
        CUDA_CHECK(hipEventElapsedTime(&stats.to_z_major_time, start_kernel, stop_kernel));
        
        // z_sweep
        CUDA_CHECK(hipEventRecord(start_kernel));
        size_t shared_size = blockSize.x * blockSize.y * sizeof(real_t);
		z_sweep<<<gridSizeZ, blockSize, shared_size>>>(d_B, nx, ny, nz, d_eps);
        CUDA_CHECK(hipEventRecord(stop_kernel));
        CUDA_CHECK(hipEventSynchronize(stop_kernel));
        CUDA_CHECK(hipEventElapsedTime(&stats.z_sweep_time, start_kernel, stop_kernel));
        
        // reorder_from_z_major
        CUDA_CHECK(hipEventRecord(start_kernel));
        reorder_from_z_major<<<gridSizeZ, blockSize>>>(d_B, d_A, nx, ny, nz);
        CUDA_CHECK(hipEventRecord(stop_kernel));
        CUDA_CHECK(hipEventSynchronize(stop_kernel));
        CUDA_CHECK(hipEventElapsedTime(&stats.from_z_major_time, start_kernel, stop_kernel));
        
        CUDA_CHECK(hipMemcpy(&h_eps, d_eps, sizeof(real_t), hipMemcpyDeviceToHost));
        
        printf(" IT = %4i   EPS = " EPS_FORMAT "\n", it, h_eps);
        if (h_eps < MAXEPS) break;
    }
	
	stats.total_time = stats.x_sweep_time + 
					   stats.y_sweep_time + 
					   stats.to_z_major_time + 
					   stats.z_sweep_time + 
					   stats.from_z_major_time;
    
    std::string profile_dir = "profiles";
    system(("mkdir -p " + profile_dir).c_str());
    std::string profile_file = profile_dir + "/gpu_profile.txt";
    write_kernel_stats(stats, profile_file);
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    printf(" ADI Benchmark Completed.\n");
    printf(" Size            = %4d x %4d x %4d\n", nx, ny, nz);
    printf(" Iterations      =       %12d\n", ITMAX);
    printf(" Time in seconds =       %12.2f\n", milliseconds / 1000.0f);
    printf(" Operation type  =     %s\n", (sizeof(real_t) == sizeof(double)) ? "double" : "float");
    printf(" GPU Memory used =     %.2f MB\n", (2.0 * nx * ny * nz * sizeof(real_t)) / (1024.0 * 1024.0));
    printf(" Block size      =     %d x %d\n", blockSize.x, blockSize.y);
    printf(" END OF ADI Benchmark\n");
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_eps));
    free(h_A);

    return 0;
}