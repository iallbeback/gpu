#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <assert.h>

#ifdef USE_REAL_DOUBLE
typedef double real_t;
#else
typedef float real_t;
#endif

#define MAXEPS 0.5
#define EPS_FORMAT "%14.7E"

#ifndef L
#define L 900
#endif

#ifndef ITMAX
#define ITMAX 20
#endif

#ifndef BLOCK_SIZE_X
#define BLOCK_SIZE_X 16
#endif

#ifndef BLOCK_SIZE_Y
#define BLOCK_SIZE_Y 4
#endif

#ifndef BLOCK_SIZE_Z
#define BLOCK_SIZE_Z 4
#endif

#define CUDA_CHECK(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

template <typename T>
__device__ static T MyatomicMax(T* address, T val);

template <typename T>
__device__ T MyatomicMax(T* address, T val) {
    if (sizeof(T) == sizeof(double)) {
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(max(val, __longlong_as_double(assumed))));
        } while (assumed != old);
        return __longlong_as_double(old);
    } else {
        int* address_as_int = (int*)address;
        int old = *address_as_int, assumed;
        do {
            assumed = old;
            old = atomicCAS(address_as_int, assumed,
                __float_as_int(fmaxf(val, __int_as_float(assumed))));
        } while (assumed != old);
        return __int_as_float(old);
    }
}

__global__ void compute_eps_and_update(real_t* __restrict A, const real_t* __restrict B, real_t* eps, int size) {
    extern __shared__ real_t shared_eps[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    int tid = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    
    real_t local_eps = 0.0;
    
    if (i > 0 && i < size - 1 && j > 0 && j < size - 1 && k > 0 && k < size - 1) {
        int idx = (k * size + j) * size + i;
        real_t tmp = fabs(B[idx] - A[idx]);
        local_eps = tmp;
        A[idx] = B[idx];
    }
    
    shared_eps[tid] = local_eps;
    __syncthreads();

    for (int s = blockDim.x * blockDim.y * blockDim.z / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_eps[tid] = fmax(shared_eps[tid], shared_eps[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        MyatomicMax(eps, shared_eps[0]);
    }
}

__global__ void update_B(const real_t* __restrict A, real_t* __restrict B, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i > 0 && i < size - 1 && j > 0 && j < size - 1 && k > 0 && k < size - 1) {
        int idx = (k * size + j) * size + i;
        int stride = size * size;
        
        real_t sum = A[idx - stride] +  // (i-1,j,k)
                    A[idx - size] +     // (i,j-1,k)
                    A[idx - 1] +        // (i,j,k-1)
                    A[idx + 1] +        // (i,j,k+1)
                    A[idx + size] +     // (i,j+1,k)
                    A[idx + stride];    // (i+1,j,k)
        
        B[idx] = sum / 6.0f;
    }
}

void print_gpu_info() {
    hipDeviceProp_t prop;
    int device;
    CUDA_CHECK(hipGetDevice(&device));
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    
    printf("GPU Device: %s\n", prop.name);
    printf("Total Global Memory: %.2f MB\n", prop.totalGlobalMem / (1024.0 * 1024.0));
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads Dim: %d x %d x %d\n", 
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
}

int main(int argc, char** argv) {
	
	dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	
	if (argc > 3) {
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
		blockSize.z = atoi(argv[3]);
		printf("Using custom block size: %d x %d x %d\n", blockSize.x, blockSize.y, blockSize.z);
	}
	
    print_gpu_info();
    
    size_t size = L * L * L * sizeof(real_t);
    printf("Using array size: %d x %d x %d\n", L, L, L);
    printf("Memory used per array: %.2f MB\n", (L*L*L*sizeof(real_t)) / (1024.0*1024.0));
    
    real_t *h_A = (real_t*)malloc(size);
    real_t *h_B = (real_t*)malloc(size);
    
    if (!h_A || !h_B) {
        printf("Error: Memory allocation failed!\n");
        return 1;
    }
    
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < L; j++) {
            for (int k = 0; k < L; k++) {
                h_A[(i * L + j) * L + k] = 0;
                if (i == 0 || j == 0 || k == 0 || i == L - 1 || j == L - 1 || k == L - 1) {
                    h_B[(i * L + j) * L + k] = 0;
                } else {
                    h_B[(i * L + j) * L + k] = 4 + i + j + k;
                }
            }
        }
    }
    
    real_t *d_A, *d_B, *d_eps;
    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_eps, sizeof(real_t)));
    
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    
    dim3 gridSize(
        (L + blockSize.x - 1) / blockSize.x,
        (L + blockSize.y - 1) / blockSize.y,
        (L + blockSize.z - 1) / blockSize.z
    );

    size_t shared_mem_size = blockSize.x * blockSize.y * blockSize.z * sizeof(real_t);
    printf("Using block size: %d x %d x %d\n", blockSize.x, blockSize.y, blockSize.z);
    printf("Shared memory per block: %zu bytes\n", shared_mem_size);
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));
    
    for (int it = 1; it <= ITMAX; it++) {
        real_t h_eps = 0;
        CUDA_CHECK(hipMemcpy(d_eps, &h_eps, sizeof(real_t), hipMemcpyHostToDevice));
        
        compute_eps_and_update<<<gridSize, blockSize, shared_mem_size>>>(d_A, d_B, d_eps, L);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        update_B<<<gridSize, blockSize>>>(d_A, d_B, L);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        CUDA_CHECK(hipMemcpy(&h_eps, d_eps, sizeof(real_t), hipMemcpyDeviceToHost));
        
        printf(" IT = %4i   EPS = " EPS_FORMAT "\n", it, h_eps);
        if (h_eps < MAXEPS) break;
    }
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    printf(" Jacobi3D GPU Benchmark Completed.\n");
    printf(" Size            = %4d x %4d x %4d\n", L, L, L);
    printf(" Iterations      =       %12d\n", ITMAX);
    printf(" Time in seconds =       %12.2f\n", milliseconds / 1000.0f);
    printf(" Operation type  =     %s\n", (sizeof(real_t) == sizeof(double)) ? "double" : "float");
    printf(" GPU Memory used =     %.2f MB\n", (2.0 * L * L * L * sizeof(real_t)) / (1024.0 * 1024.0));
    printf(" Block size      =     %d x %d x %d\n", blockSize.x, blockSize.y, blockSize.z);
    printf(" END OF Jacobi3D Benchmark\n");
        
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_eps));
    free(h_A);
    free(h_B);
    
    return 0;
}